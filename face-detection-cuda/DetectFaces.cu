#include "DetectFaces.hpp"


#define NUM_SCALES 1


void detectFaces(float *intImage_h,
                 int rows,
                 int columns,
                 int stride,
                 int *subWinOffsets_h,
                 int subWinNum,
                 int subWinSize)
{
   // Kernel setup
   int blocksPerGrid = 1;
   int threadPerBlock = 64;
   int threads = blocksPerGrid * threadPerBlock;


   // Copying integral image to device
   float *intImage_d;
   // Size of integral image in bytes
   size_t intImageSize = rows * columns * sizeof(float);
   cudaCheckError(hipMalloc(&intImage_d, intImageSize));
   cudaCheckError(hipMemcpy(intImage_d, intImage_h, intImageSize, hipMemcpyHostToDevice));


   // Copying window offsets to device
   int *subWinOffsets_d;
   int nValidSubWindows = subWinNum;
   // Size of sub-window offsets in bytes
   size_t subWinOffsetsSize = nValidSubWindows * sizeof(int);
   cudaCheckError(hipMalloc(&subWinOffsets_d, subWinOffsetsSize));
   cudaCheckError(hipMemcpy(subWinOffsets_d, subWinOffsets_h, subWinOffsetsSize, hipMemcpyHostToDevice));


   // Init array for face detected,
   // containg 1 if face detected, else 0 - face not detected
   int *faceDetected_d;
   cudaCheckError(hipMalloc(&faceDetected_d, nValidSubWindows * sizeof(int)));
   cudaCheckError(hipMemset(faceDetected_d, 0, nValidSubWindows * sizeof(int));


   // Array to hold maximum feature value for each sub window
   // for debugging
   float *results_d;
   cudaCheckError(hipMalloc(&results_d, nValidSubWindows * sizeof(float)));
   cudaCheckError(hipMemset(results_d, 0, nValidSubWindows * sizeof(float)));



}

