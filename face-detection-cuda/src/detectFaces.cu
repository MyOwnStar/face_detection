#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <thrust/remove.h>
#include <thrust/device_ptr.h>

#include "kernels/ID1.cu"
#include "kernels/ID2.cu"
#include "kernels/ID3.cu"
#include "kernels/ID4.cu"

#include "cudaCheckError.cu"

#define TH_PER_BLOCK 64
#define N_SCALES      1


void debugResults(int *facesDetected_d, float *results_d, int nValidSubWindows);

int compact(int *winOffsets_d, int *faceDetected_d, int nValidSubWindows);

void kernel_heading(char *heading, int blocks, int th_per_block, int threads, int nValidSubWindows);

void kernel_footer(char *msg, clock_t kernel_start);


void cuda_detect_faces(float *hostIntImage,
                       int rows,
                       int columns,
                       int stride,
                       int *hostSubWinOffsets,
                       int subWinNum,
                       int subWinSize)
{

   // Initialize kernel size --------------------------------------------------
   int blocksPerGrid = 1;
   int threadPerBlock = TH_PER_BLOCK;
   int threads = blocksPerGrid * threadPerBlock;


   // Initialize clock --------------------------------------------------------
   clock_t test_start = clock();
   clock_t kernel_start;


   // Copy Integral Image to device -------------------------------------------
   float *intImg_d;
   hipMalloc(&intImg_d, rows * columns * sizeof(float));
   hipMemcpy(intImg_d, hostIntImage, rows * columns * sizeof(float), hipMemcpyHostToDevice);


   // Copy window mOffsets to device -------------------------------------------
   int *winOffsets_d;
   int nValidSubWindows = subWinNum;

   hipMalloc(&winOffsets_d, nValidSubWindows * sizeof(int));
   hipMemcpy(winOffsets_d, hostSubWinOffsets, nValidSubWindows * sizeof(int), hipMemcpyHostToDevice);


   // Initialize device 'boolean' face detected array -------------------------
   int *faceDetected_d;
   hipMalloc(&faceDetected_d, nValidSubWindows * sizeof(int));
   hipMemset(faceDetected_d, 0, nValidSubWindows * sizeof(int));


   // Initialize results array for debugging... -------------------------------
   float *results_d;
   hipMalloc(&results_d, nValidSubWindows * sizeof(float));
   hipMemset(results_d, 0, nValidSubWindows * sizeof(float));



   //==========================================================================
   // Run ID1 -----------------------------------------------------------------
   kernel_heading("ID1", blocksPerGrid, threadPerBlock, threads, nValidSubWindows);
   kernel_start = clock();
   for (int i = 2; i < 2 + N_SCALES; ++i)
   {
      ID1kernel << < blocksPerGrid, threadPerBlock >> > (intImg_d,               // Itegral Image
              stride,                  //	Stride
              winOffsets_d,            //	Sub-Window Offsets
              subWinSize,               //	Sub-Window Size
              nValidSubWindows,      //	Number of Sub Windows
              subWinSize / (5 * (i)),         // Scale of the feature
              faceDetected_d,         //	Array to hold if a face was detected
              results_d               //	Array to hold maximum feature value for each sub window
      );
   }
   kernel_footer("ID1", kernel_start);
   debugResults(faceDetected_d, results_d, nValidSubWindows);

   // Compact -----------------------------------------------------------------
   nValidSubWindows = compact(winOffsets_d, faceDetected_d, nValidSubWindows);

   // Prepare for next run ----------------------------------------------------
   hipMemset(faceDetected_d, 0, nValidSubWindows * sizeof(float));
   hipMemset(results_d, 0, nValidSubWindows * sizeof(float));



   //==========================================================================
   // Run ID2 -----------------------------------------------------------------
   kernel_heading("ID2", blocksPerGrid, threadPerBlock, threads, nValidSubWindows);
   kernel_start = clock();
   for (int i = 2; i < 2 + N_SCALES; ++i)
   {
      ID2kernel << < blocksPerGrid, threadPerBlock >> > (intImg_d,               // Itegral Image
              stride,                  //	Stride
              winOffsets_d,            //	Sub-Window Offsets
              subWinSize,               //	Sub-Window Size
              nValidSubWindows,      //	Number of Sub Windows
              subWinSize / (5 * (i)),         // Scale of the feature
              faceDetected_d,         //	Array to hold if a face was detected
              results_d               //	Array to hold maximum feature value for each sub window
      );
   }
   kernel_footer("ID2", kernel_start);
   debugResults(faceDetected_d, results_d, nValidSubWindows);

   // Compact -----------------------------------------------------------------
   nValidSubWindows = compact(winOffsets_d, faceDetected_d, nValidSubWindows);

   // Prepare for next run ----------------------------------------------------
   hipMemset(faceDetected_d, 0, nValidSubWindows * sizeof(float));
   hipMemset(results_d, 0, nValidSubWindows * sizeof(float));



   //==========================================================================
   // Run ID3 -----------------------------------------------------------------
   kernel_heading("ID3", blocksPerGrid, threadPerBlock, threads, nValidSubWindows);
   kernel_start = clock();
   for (int i = 2; i < 2 + N_SCALES; ++i)
   {
      ID3kernel << < blocksPerGrid, threadPerBlock >> > (intImg_d,               // Itegral Image
              stride,                  //	Stride
              winOffsets_d,            //	Sub-Window Offsets
              subWinSize,               //	Sub-Window Size
              nValidSubWindows,      //	Number of Sub Windows
              subWinSize / (5 * (i)),         // Scale of the feature
              faceDetected_d,         //	Array to hold if a face was detected
              results_d               //	Array to hold maximum feature value for each sub window
      );
   }
   kernel_footer("ID3", kernel_start);
   debugResults(faceDetected_d, results_d, nValidSubWindows);

   // Compact -----------------------------------------------------------------
   nValidSubWindows = compact(winOffsets_d, faceDetected_d, nValidSubWindows);

   // Prepare for next run ----------------------------------------------------
   hipMemset(faceDetected_d, 0, nValidSubWindows * sizeof(float));
   hipMemset(results_d, 0, nValidSubWindows * sizeof(float));



   //==========================================================================
   // Run ID4 -----------------------------------------------------------------
   kernel_heading("ID4", blocksPerGrid, threadPerBlock, threads, nValidSubWindows);
   kernel_start = clock();
   for (int i = 2; i < 2 + N_SCALES; ++i)
   {
      ID4kernel << < blocksPerGrid, threadPerBlock >> > (intImg_d,               // Itegral Image
              stride,                  //	Stride
              winOffsets_d,            //	Sub-Window Offsets
              subWinSize,               //	Sub-Window Size
              nValidSubWindows,      //	Number of Sub Windows
              subWinSize / (5 * (i)),         // Scale of the feature
              faceDetected_d,         //	Array to hold if a face was detected
              results_d              //	Array to hold maximum feature value for each sub window
      );
   }
   kernel_footer("ID4", kernel_start);
   debugResults(faceDetected_d, results_d, nValidSubWindows);

   // Compact -----------------------------------------------------------------
   nValidSubWindows = compact(winOffsets_d, faceDetected_d, nValidSubWindows);


   //==========================================================================
   // Test Results ------------------------------------------------------------
   printf("Face Results\n\n");
   printf("Completed test in %f seconds\n", ((double) clock() - test_start) / CLOCKS_PER_SEC);
   if (nValidSubWindows > 0)
   {
      printf("A face was detected\n");
   }



   // Cleanup -----------------------------------------------------------------
   hipFree(intImg_d);
   hipFree(winOffsets_d);
   hipFree(faceDetected_d);
   hipFree(results_d);
}


void debugResults(int *facesDetected_d, float *results_d, int nValidSubWindows)
{
   int *facesDetected = (int *) malloc(nValidSubWindows * sizeof(int));
   float *results = (float *) malloc(nValidSubWindows * sizeof(float));

   hipMemcpy(facesDetected, facesDetected_d, nValidSubWindows * sizeof(int), hipMemcpyDeviceToHost);
   hipMemcpy(results, results_d, nValidSubWindows * sizeof(float), hipMemcpyDeviceToHost);


   for (int i = 0; i < nValidSubWindows; ++i)
   {
      printf("%4d - %f: ", i, results[i]);
      if (facesDetected[i] == 0)
      {
      } else if (facesDetected[i] == 1)
      {
         printf(" FACE DETECTED");
      } else
      {
         printf(" Well poo!!!");
      }
      printf("\n");
   }

   free(facesDetected);
   free(results);
}


int compact(int *winOffsets_d, int *faceDetected_d, int nValidSubWindows)
{
   clock_t clk = clock();

   // Cast to thrust device pointers
   thrust::device_ptr<int> offsets_ptr(winOffsets_d);
   thrust::device_ptr<int> detected_ptr(faceDetected_d);

   // Perform the compact!
   thrust::device_ptr<int> new_end = thrust::remove_if(offsets_ptr, offsets_ptr + nValidSubWindows, detected_ptr,
                                                       thrust::logical_not<int>());

   // Compute the length of compacted array
   int len = new_end - offsets_ptr;

   printf("Compacting completed in %f seconds\n", ((double) clock() - clk) / CLOCKS_PER_SEC);

   printf("Possible faces: %d\n\n", len);

   // Return the length of the compacted array
   return len;
}


void kernel_heading(char *heading, int blocks, int th_per_block, int threads, int nValidSubWindows)
{
   printf("\n");
   printf("Running %s --------\n", heading);
   printf("Blocks:   %d\n", blocks);
   printf("Th/Block: %d\n", th_per_block);
   printf("Threads:  %d\n", threads);
   printf("Windows:  %d\n", nValidSubWindows);
}

void kernel_footer(char *msg, clock_t kernel_start)
{
   hipDeviceSynchronize();
   printf("%s completed in %f seconds\n", msg, ((double) clock() - kernel_start) / CLOCKS_PER_SEC);
}