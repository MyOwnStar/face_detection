#include "hip/hip_runtime.h"
#include "IntegralImage.hpp"

#define THREADS_PER_BLOCK 384

__global__
void rowsScan(float *data, int rows, int columns, size_t stride)
{
   int row = blockIdx.x * blockDim.x + threadIdx.x;

   if(row < rows)
   {
      for(int col = 1; col < columns; ++col)
      {
         data[row * stride + col] = data[row * stride + col] + data[row * stride + col - 1];
      }
   }
}

__global__
void colsScan(float *data, int rows, int columns, size_t stride)
{
   int col = blockIdx.x * blockDim.x + threadIdx.x;

   if(col < columns)
   {
      for(int row = 1; row < rows; ++row)
      {
         data[row * stride + col] = data[row * stride + col] + data[(row - 1) * stride + col];
      }
   }
}

void gpuIntImage(float *hostData, int rows, int columns, size_t stride)
{
   size_t sizeInByte = rows * columns * sizeof(float);

   float *deviceData;

   cudaCheckError(hipMalloc(&deviceData, sizeInByte));

   cudaCheckError(hipMemcpy(deviceData, hostData, sizeInByte, hipMemcpyHostToDevice));

   int numBlocks = rows / THREADS_PER_BLOCK;

   //std::cout << "numBlocks: " << numBlocks << std::endl;

   const clock_t start = clock();
   rowsScan<<<numBlocks, THREADS_PER_BLOCK>>>(deviceData, rows, columns, stride);
   cudaCheckError(hipPeekAtLastError());
   cudaCheckError(hipDeviceSynchronize());

   numBlocks = columns / THREADS_PER_BLOCK + 1;

   //std::cout << "numBlocks: " << numBlocks << std::endl;

   cudaCheckError(hipDeviceSynchronize());

   colsScan<<<numBlocks, THREADS_PER_BLOCK>>>(deviceData, rows, columns, stride);
   cudaCheckError(hipPeekAtLastError());
   cudaCheckError(hipDeviceSynchronize());

   cudaCheckError(hipDeviceSynchronize());
   std::cout << "CUDA Intagral Image: " << static_cast<float>(clock() - start) << " ms" << std::endl;

   cudaCheckError(hipMemcpy(hostData, deviceData, sizeInByte, hipMemcpyDeviceToHost));

   cudaCheckError(hipFree(deviceData));
}